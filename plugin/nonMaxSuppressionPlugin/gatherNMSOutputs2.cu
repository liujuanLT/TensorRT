#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "plugin.h"
#include "hip/hip_fp16.h"
#include "gatherNMSOutputs2.h"
#include <array>

// __half minus with fallback to float for old sm
inline __device__ __half minus_fb(const __half & a, const __half & b) {
#if __CUDA_ARCH__ >= 530
    return a - b;
#else
    return __float2half(__half2float(a) - __half2float(b));
#endif
}

template <typename T_BBOX>
__device__ T_BBOX saturate2(T_BBOX v)
{
    return max(min(v, T_BBOX(1)), T_BBOX(0));
}

template <>
__device__ __half saturate2(__half v)
{
#if __CUDA_ARCH__ >= 800
    return __hmax(__hmin(v, __half(1)), __half(0));
#elif __CUDA_ARCH__ >= 530
    return __hge(v, __half(1)) ? __half(1) : (__hle(v, __half(0)) ? __half(0) : v);
#else
    return max(min(v, float(1)), float(0));
#endif
}

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void gatherNMSOutputs2_kernel(
        const bool shareLocation,
        const int numImages,
        const int numPredsPerClass,
        const int numClasses,
        const int topK,
        const int keepTopK,
        const int* indices,
        const T_SCORE* scores,
        const T_BBOX* bboxData,
        int* nmsedResult,
        bool clipBoxes,
        const T_SCORE scoreShift
        )
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x;
         i < numImages * keepTopK;
         i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId]; 
        //const T_SCORE score = scores[offset + detId];
        if (index == -1)
        {
            nmsedResult[i] = -1;
            nmsedResult[i+1] = -1;
            nmsedResult[i+2] = -1;
        }
        else
        {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + bboxOffset) * 4;
            const int classId = (index % (numClasses * numPredsPerClass)) / numPredsPerClass; // label
            nmsedResult[i] = imgId;
            nmsedResult[i+1] = classId;
            nmsedResult[i+2] = bboxId;
           
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSOutputs2_gpu(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* nmsedResult,
    bool clipBoxes,
    const float scoreShift
    )
{
    //hipMemsetAsync(numDetections, 0, numImages * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSOutputs2_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
                                                                           numClasses, topK, keepTopK,
                                                                           (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData,
                                                                           (int*) nmsedResult,
                                                                           clipBoxes,
                                                                           T_SCORE(scoreShift)
                                                                            );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSOutputs2 LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t,
                               const bool,
                               const int,
                               const int,
                               const int,
                               const int,
                               const int,
                               const void*,
                               const void*,
                               const void*,
                               void*,
                               bool,
                               const float);
struct nmsOutLaunchConfig
{
    DataType t_bbox;
    DataType t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DataType t_bbox, DataType t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::array<nmsOutLaunchConfig, 2> nmsOutLCOptions = {
  nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherNMSOutputs2_gpu<float, float>),
  nmsOutLaunchConfig(DataType::kHALF, DataType::kHALF, gatherNMSOutputs2_gpu<__half, __half>)
};

pluginStatus_t gatherNMSOutputs2(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const DataType DT_BBOX,
    const DataType DT_SCORE,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* nmsedResult,
    bool clipBoxes,
    const float scoreShift
    )
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutLCOptions.size(); ++i)
    {
        if (lc == nmsOutLCOptions[i])
        {
            DEBUG_PRINTF("gatherNMSOutputs2 kernel %d\n", i);
            return nmsOutLCOptions[i].function(stream,
                                          shareLocation,
                                          numImages,
                                          numPredsPerClass,
                                          numClasses,
                                          topK,
                                          keepTopK,
                                          indices,
                                          scores,
                                          bboxData,
                                          nmsedResult,
                                          clipBoxes,
                                          scoreShift
                                          );
        }
    }
    return STATUS_BAD_PARAM;
}
